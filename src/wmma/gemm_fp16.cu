#include "hip/hip_runtime.h"
#include "wmma_function.h"
using namespace nvcuda;

template <int fragment_m, int fragment_n, int fragment_k>
__global__ void wmma_fp16_kernel(const half *a, const half *b, int M, int N,
                                 float *c, int K) {
  size_t row = fragment_m * blockIdx.y;
  size_t col = fragment_n * blockIdx.x;

  if (row >= M || col >= N) {
    return;
  }

  wmma::fragment<wmma::accumulator, fragment_m, fragment_n, fragment_k, float>
      c_frag;
  wmma::fill_fragment(c_frag, 0.0f);
  size_t loop_n = (K + fragment_k - 1) / fragment_k;
#pragma unroll
  for (size_t i = 0; i < loop_n; ++i) {
    wmma::fragment<wmma::matrix_a, fragment_m, fragment_n, fragment_k, half,
                   wmma::row_major>
        a_frag;
    wmma::fragment<wmma::matrix_b, fragment_m, fragment_n, fragment_k, half,
                   wmma::col_major>
        b_frag;
    wmma::load_matrix_sync(a_frag, a + row * K + i * fragment_k, K);
    wmma::load_matrix_sync(b_frag, b + col * K + i * fragment_k, K);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  }
  wmma::store_matrix_sync(c + row * N + col, c_frag, N, wmma::mem_row_major);
}

template <int fragment_m, int fragment_n, int fragment_k>
__global__ void wmma_fp16_kernel(const half *a, const half *b, int M, int N,
                                 half *c, int K) {
  size_t row = fragment_m * blockIdx.y;
  size_t col = fragment_n * blockIdx.x;

  if (row >= M || col >= N) {
    return;
  }

  wmma::fragment<wmma::accumulator, fragment_m, fragment_n, fragment_k, half>
      c_frag;
  wmma::fill_fragment(c_frag, 0.0f);
  size_t loop_n = (K + fragment_k - 1) / fragment_k;
#pragma unroll
  for (size_t i = 0; i < loop_n; ++i) {
    wmma::fragment<wmma::matrix_a, fragment_m, fragment_n, fragment_k, half,
                   wmma::row_major>
        a_frag;
    wmma::fragment<wmma::matrix_b, fragment_m, fragment_n, fragment_k, half,
                   wmma::col_major>
        b_frag;
    wmma::load_matrix_sync(a_frag, a + row * K + i * fragment_k, K);
    wmma::load_matrix_sync(b_frag, b + col * K + i * fragment_k, K);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  }
  wmma::store_matrix_sync(c + row * N + col, c_frag, N, wmma::mem_row_major);
}

template <typename SRC_A, typename SRC_B, typename DST_C>
int wmma_fp16_imp(const SRC_A *a, const SRC_B *b, int M, int N, DST_C *c, int K,
                  bool matrix_b_is_col_major,
                  FP16TensorKenerlType kenerl_type) {
  if (M < 0 || N < 0 || K < 0) {
    return -1;
  }
  int kenerl_m = 0, kenerl_n = 0, kenerl_k = 0;
  switch (kenerl_type) {
    case FP16TensorKenerlType::TensorKenerl_16_16_16:
      kenerl_m = 16;
      kenerl_n = 16;
      kenerl_k = 16;
      break;
    case FP16TensorKenerlType::TensorKenerl_8_32_16:
      kenerl_m = 8;
      kenerl_n = 32;
      kenerl_k = 16;
      break;
    case FP16TensorKenerlType::TensorKenerl_32_8_16:
      kenerl_m = 32;
      kenerl_n = 8;
      kenerl_k = 16;
      break;
  }
  if (kenerl_m == 0 || kenerl_n == 0 || kenerl_k == 0) {
    return -1;
  }
  dim3 grid;
  int m;
  int n;
  int k;
  half *dev_a;
  half *dev_b;
  DST_C *dev_c;
  data_preprocess(grid, a, b, M, N, K, dev_a, dev_b, dev_c, m, n, k, kenerl_m,
                  kenerl_n, kenerl_k, matrix_b_is_col_major);
  switch (kenerl_type) {
    case FP16TensorKenerlType::TensorKenerl_16_16_16:
      // wmma_fp16_kernel_16_16_16<<<grid, WARP_SIZE>>>(dev_a, dev_b, m, n,
      // dev_c, k);
      wmma_fp16_kernel<16, 16, 16>
          <<<grid, WARP_SIZE>>>(dev_a, dev_b, m, n, dev_c, k);
      break;
    case FP16TensorKenerlType::TensorKenerl_8_32_16:
      // wmma_fp16_kernel_8_32_16<<<grid, WARP_SIZE>>>(dev_a, dev_b, m, n,
      // dev_c, k);
      wmma_fp16_kernel<8, 32, 16>
          <<<grid, WARP_SIZE>>>(dev_a, dev_b, m, n, dev_c, k);
      break;
    case FP16TensorKenerlType::TensorKenerl_32_8_16:
      // wmma_fp16_kernel_32_8_16<<<grid, WARP_SIZE>>>(dev_a, dev_b, m, n,
      // dev_c, k);
      wmma_fp16_kernel<32, 8, 16>
          <<<grid, WARP_SIZE>>>(dev_a, dev_b, m, n, dev_c, k);
      break;
  }

  for (int i = 0; i < M; ++i) {
    CUDA_CHECK(hipMemcpy(c + i * N, dev_c + i * n, N * sizeof(DST_C),
                          hipMemcpyDeviceToHost));
  }

  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipFree(dev_b));
  CUDA_CHECK(hipFree(dev_c));
  return 0;
}

int wmma_fp16(const float *a, const float *b, int M, int N, float *c, int K,
              bool matrix_b_is_col_major, FP16TensorKenerlType kenerl_type) {
  return wmma_fp16_imp(a, b, M, N, c, K, matrix_b_is_col_major, kenerl_type);
}

int wmma_fp16(const half *a, const half *b, int M, int N, float *c, int K,
              bool matrix_b_is_col_major, FP16TensorKenerlType kenerl_type) {
  return wmma_fp16_imp(a, b, M, N, c, K, matrix_b_is_col_major, kenerl_type);
}

int wmma_fp16(const half *a, const half *b, int M, int N, half *c, int K,
              bool matrix_b_is_col_major, FP16TensorKenerlType kenerl_type) {
  return wmma_fp16_imp(a, b, M, N, c, K, matrix_b_is_col_major, kenerl_type);
}
